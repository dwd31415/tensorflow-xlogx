#include "hip/hip_runtime.h"
#include "settings.h"

__global__ void cuda_op_function(const float *in, const int N, float* out){
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
         i += blockDim.x * gridDim.x) {
        if(in[i] == 0.0f) {
            out[i] = 0.0f;
        } else {
            out[i] = logf(in[i]) * in[i];
        }
    }
}

bool cuda_op_launcher(const float *in, const int N, float* out){
    hipGetLastError();
    cuda_op_function<<<32,256>>>(in, N, out);
    if (hipGetLastError() != hipSuccess)
    {
        return false;
    }else{
        return true;
    }
}

__global__ void cuda_op_gradient(const float *in, const int N, float* out){
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
         i += blockDim.x * gridDim.x) {
#if FIX_DERIVATIVE_FROM_0_TO_EPSILON
        if (in[i] < EPSILON){
#else
        if (in[i] == 0){
#endif
            out[i] = FIXED_DERIVATIVE_VALUE;
        } else {
            out[i] = 1 + logf(in[i]);
        }
    }
}

bool cuda_grad_op_launcher(const float *in, const int N, float* out){
    hipGetLastError();
    cuda_op_gradient<<<32,256>>>(in, N, out);
    if (hipGetLastError() != hipSuccess)
    {
        return false;
    }else{
        return true;
    }
}
